#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <string.h>

extern "C"{
  #include "matrix.h"
};

#define CUDA_CHECK_ERROR() do { \
    hipError_t err = hipGetLastError(); \
    if (err != hipSuccess) { \
        printf("CUDA error: %s at line %d\n", hipGetErrorString(err), __LINE__); \
        exit(err); \
    } \
} while(0)

int main(void) {
    Matrix inputMatrix;
    Matrix identityMatrix;
    int size;

    inputMatrix = readMatrixFromFile();
    identityMatrix = createIdentityMatrix(inputMatrix.col);
    size = inputMatrix.col;
    bool invertible = true;

    
    printf("Input matrix:\n");
    for (size_t i=0; i<size; i++){
        for (size_t j=0; j<size; j++){
            printf("%.4f\t", inputMatrix.buffer[i * size + j]);
        }
        printf("\n");
    }
    printf("Identity matrix:\n");
    for (size_t i=0; i<size; i++){
        for (size_t j=0; j<size; j++){
            printf("%.4f\t", identityMatrix.buffer[i * size + j]);
        }
        printf("\n");
    }

    dim3 block(256,1,1);
    dim3 grid((size + block.x - 1) / block.x, 1, 1);

    // GPU allocation memory
    double *d_inputMatrix;
    double *d_identityMatrix;
    bool *d_invertible;
    int *d_size;

    size_t free_bytes, total_bytes;
    hipMemGetInfo(&free_bytes, &total_bytes);
    printf("Available device memory: %zu bytes\n", free_bytes);
    printf("Total device memory: %zu bytes\n", total_bytes);

    hipMalloc((void **)&d_inputMatrix, size * size * sizeof(double));
    CUDA_CHECK_ERROR();

    hipMalloc((void **)&d_identityMatrix, size * size * sizeof(double));
    CUDA_CHECK_ERROR();

    hipMalloc((void **)&d_invertible, sizeof(bool));
    CUDA_CHECK_ERROR();

    hipMalloc((void **)&d_size, sizeof(int));
    CUDA_CHECK_ERROR();

    hipMemcpy(d_inputMatrix, inputMatrix.buffer, size * size * sizeof(double), hipMemcpyHostToDevice);
    CUDA_CHECK_ERROR();

    hipMemcpy(d_identityMatrix, identityMatrix.buffer, size * size * sizeof(double), hipMemcpyHostToDevice);
    CUDA_CHECK_ERROR();

    hipMemcpy(d_invertible, &invertible, sizeof(bool), hipMemcpyHostToDevice);
    CUDA_CHECK_ERROR();

    hipMemcpy(d_size, &size, sizeof(int), hipMemcpyHostToDevice);
    CUDA_CHECK_ERROR();

    printf("Input matrix:\n");
    double *inputMatrixHost = (double *)malloc(size * size * sizeof(double));
    hipMemcpy(inputMatrixHost, d_inputMatrix, size * size * sizeof(double), hipMemcpyDeviceToHost);
    for (size_t i = 0; i < size; i++) {
        for (size_t j = 0; j < size; j++) {
            printf("%.4f\t", inputMatrixHost[i * size + j]);
        }
        printf("\n");
    }
    free(inputMatrixHost);

    printf("Identity matrix:\n");
    double *identityMatrixHost = (double *)malloc(size * size * sizeof(double));
    hipMemcpy(identityMatrixHost, d_identityMatrix, size * size * sizeof(double), hipMemcpyDeviceToHost);
    for (size_t i = 0; i < size; i++) {
        for (size_t j = 0; j < size; j++) {
            printf("%.4f\t", identityMatrixHost[i * size + j]);
        }
        printf("\n");
    }
    free(identityMatrixHost);

    hipFree(d_inputMatrix);
    hipFree(d_identityMatrix);
    hipFree(d_invertible);
    hipFree(d_size);

    return 0;
}