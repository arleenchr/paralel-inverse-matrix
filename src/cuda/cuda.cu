#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <string.h>

extern "C"{
  #include "matrix.h"
};

#define CUDA_CHECK_ERROR() do { \
    hipError_t err = hipGetLastError(); \
    if (err != hipSuccess) { \
        printf("CUDA error: %s at line %d\n", hipGetErrorString(err), __LINE__); \
        exit(err); \
    } \
} while(0)

/*Getter: row in a certain index*/
double* getRow (Matrix m, size_t rowNum){
    double* row = (double *)calloc(m.col, sizeof(double));
    memcpy(row, &(m.buffer[rowNum * m.row]), m.row * sizeof(double));
    return row;
}

/*Swap row with another row*/
void swapRow(Matrix* matrix, int row1, int row2){
    if(row1 < 0 || row1 >= matrix->row || row2 < 0 || row2 >= matrix->row){
        fprintf(stderr, "Invalid row indices\n");
        return;
    }

    int startIndexRow2 = row2 * matrix->col;
    int startIndexRow1 = row1 * matrix->col;

    double* temp_row = (double*)malloc(matrix->col * sizeof(double));
    if(temp_row == NULL){
        fprintf(stderr, "Memory allocation failed\n");
        return;
    }

    // Copy row1 to temp_row
    memcpy(temp_row, &(matrix->buffer[startIndexRow1]), matrix->col * sizeof(double));

    // Copy row2 to row1
    memcpy(&(matrix->buffer[startIndexRow1]), &(matrix->buffer[startIndexRow2]), matrix->col * sizeof(double));

    // Copy temp_row to row2
    memcpy(&(matrix->buffer[startIndexRow2]), temp_row, matrix->col * sizeof(double));

    free(temp_row);
}

__global__ void eliminate(double* inputMatrix, double* identityMatrix, int size, size_t it){
    size_t row = (blockIdx.x*blockDim.x) + threadIdx.x;
	size_t col = (blockIdx.y*blockDim.y) + threadIdx.y;

    if (row < size && col < size && row != it){
        identityMatrix[row * size + col] -= identityMatrix[it * size + col] * inputMatrix[row * size + it];
        
        if (col != it) {
            inputMatrix[row * size + col] -= inputMatrix[it * size + col] * inputMatrix[row * size + it];
        }
    }
}

__global__ void normalize(double* inputMatrix, double* identityMatrix, int size, size_t it){
    size_t row = (blockIdx.x*blockDim.x) + threadIdx.x;
	size_t col = (blockIdx.y*blockDim.y) + threadIdx.y;

    if (row < size && col < size && row != it && col == it) {
       inputMatrix[row * size + col] = 0;
    }
}

__global__ void reduce_nodiag(double* inputMatrix, double* identityMatrix, int size, size_t it){
    size_t row = (blockIdx.x*blockDim.x) + threadIdx.x;
	  size_t col = (blockIdx.y*blockDim.y) + threadIdx.y;
    if (row < size && col < size && row == it && row != col){
        double pivotFactor = inputMatrix[it * size + it];
        inputMatrix[it * size + col] /= pivotFactor;
        identityMatrix[it * size + col] /= pivotFactor;
    }
}

__global__ void reduce_diag(double* inputMatrix, double* identityMatrix, int size, size_t it){
    size_t row = (blockIdx.x*blockDim.x) + threadIdx.x;
	  size_t col = (blockIdx.y*blockDim.y) + threadIdx.y;
    if (row < size && col < size && row == col && row == it){
        double pivotFactor = inputMatrix[it * size + it];

        inputMatrix[it * size + col] /= pivotFactor;
        identityMatrix[it * size + col] /= pivotFactor;
    }
}

int main(void) {
    Matrix inputMatrix;
    Matrix identityMatrix;
    int size;
    
    inputMatrix = readMatrixFromFile();
    identityMatrix = createIdentityMatrix(inputMatrix.col);
    size = inputMatrix.col;
    bool invertible = true;

    dim3 block(16,16);
    int gridRow = (size+15)/16;
    int gridCol = (size+15)/16;
    dim3 grid(gridRow,gridCol);

    // GPU allocation memory
    double *d_inputMatrix;
    double *d_identityMatrix;
    bool *d_invertible;
    int *d_size;

    hipMalloc((void **)&d_inputMatrix, size * size * sizeof(double));
    CUDA_CHECK_ERROR();

    hipMalloc((void **)&d_identityMatrix, size * size * sizeof(double));
    CUDA_CHECK_ERROR();

    hipMalloc((void **)&d_invertible, sizeof(bool));
    CUDA_CHECK_ERROR();

    hipMalloc((void **)&d_size, sizeof(int));
    CUDA_CHECK_ERROR();

    for (size_t i = 0; i < size; i++){
        /* Partial Pivoting */
        /* Swapping indivisible row */
        double* colBuffer = getColFromMatrix(inputMatrix, i);

        if (colBuffer[i] == 0.){
            // Swap rows
            // search for the nearest non-zero row
            for (size_t swapIdx = i+1; swapIdx < size; swapIdx++){
                if (colBuffer[swapIdx] != 0.){
                    {
                        swapRow(&inputMatrix, i, swapIdx);
                        swapRow(&identityMatrix, i, swapIdx);
                    }
                    break;
                } else if (swapIdx == size - 1){
                    {
                        invertible = false;
                        fprintf(stderr, "Matrix can not be inversed.\n");
                    }
                }
            }
        }
        // Ensure all threads have checked invertibility before proceeding
        if (!invertible) {
            exit(1);
        }
        free(colBuffer);

        hipMemcpy(d_inputMatrix, inputMatrix.buffer, size * size * sizeof(double), hipMemcpyHostToDevice);
        CUDA_CHECK_ERROR();

        hipMemcpy(d_identityMatrix, identityMatrix.buffer, size * size * sizeof(double), hipMemcpyHostToDevice);
        CUDA_CHECK_ERROR();

        hipMemcpy(d_invertible, &invertible, sizeof(bool), hipMemcpyHostToDevice);
        CUDA_CHECK_ERROR();

        hipMemcpy(d_size, &size, sizeof(int), hipMemcpyHostToDevice);
        CUDA_CHECK_ERROR();

        reduce_nodiag<<<grid,block>>>(d_inputMatrix, d_identityMatrix, size, i);
        CUDA_CHECK_ERROR();

        reduce_diag<<<grid,block>>>(d_inputMatrix, d_identityMatrix, size, i);
        CUDA_CHECK_ERROR();

        eliminate<<<grid,block>>>(d_inputMatrix, d_identityMatrix, size, i);
        CUDA_CHECK_ERROR();

        normalize<<<grid,block>>>(d_inputMatrix, d_identityMatrix, size, i);
        CUDA_CHECK_ERROR();

        hipDeviceSynchronize();

        hipMemcpy(inputMatrix.buffer, d_inputMatrix, size * size * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(identityMatrix.buffer, d_identityMatrix, size * size * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(&invertible, d_invertible, sizeof(bool), hipMemcpyDeviceToHost);
        CUDA_CHECK_ERROR();
    }

    printf("%d\n", size);
    printMatrix(identityMatrix);

    hipFree(d_inputMatrix);
    hipFree(d_identityMatrix);
    hipFree(d_invertible);
    hipFree(d_size);

    freeMatrix(&inputMatrix);
    freeMatrix(&identityMatrix);

    return 0;
}